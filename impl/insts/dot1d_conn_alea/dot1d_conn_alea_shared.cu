#include "hip/hip_runtime.h"
#include "dot1d_conn_alea.cuh"

#define BLOQUE 16

static __global__ void kerd_stricte(
	uint _t_MODE, uint GRAINE,
	uint X_vars, uint Y_vars,
	uint X, uint Y,
	uint depart, uint T,
	uint DEPART_x,
	float * x, float * y,
	float * p,
	float * locd)
{
	// <KERD(T, BLOQUE), KERD(Y,BLOQUE)>
	// <         BLOQUE,         BLOQUE>

	__shared__ float __partage__x[BLOQUE][BLOQUE];
	__shared__ float __partage__p[BLOQUE][BLOQUE];

	uint thx = threadIdx.x;
	uint thy = threadIdx.y;

	uint _y = thx + blockIdx.x * blockDim.x;
	uint _t = thy + blockIdx.y * blockDim.y;

	float s = 0;

	FOR(0, d, CONNECTIONS/BLOQUE) {
		__partage__x[thy][thx] = x[(depart+_t)*(X_vars) + DEPART_x+ POINTEUR(_y,d*BLOQUE + thx,X)];
		__partage__p[thy][thx] = p[_y*(CONNECTIONS+1) + d*BLOQUE + thy];
		__syncthreads();

	//#pragma unroll
		FOR(0, i, BLOQUE) s += __partage__x[thy][i] * __partage__p[i][thx];
		__syncthreads();
	};

#define __partage__b __partage__x[0]

	if (thy == 0) __partage__b[thx] = p[_y*(CONNECTIONS+1) + (CONNECTIONS+1-1)];
	__syncthreads();

	s = (s + __partage__b[thx]);
	float a = dot1d_conn_alea_ACTIV(ACTIVATION, s);
	   y[/*(depart+_t)*/(0+_t)*Y + _y] = a;
	locd[/*(depart+_t)*/(0+_t)*Y + _y] = dot1d_conn_alea_dACTIV(ACTIVATION, s,a);
};

void nvidia_dot1d_conn_alea_shared(
	uint _t_MODE, uint GRAINE,
	uint X_vars, uint Y_vars,
	uint X, uint Y,
	uint depart, uint T,
	uint DEPART_x,
	float * x, float * y,
	float * p,
	float * locd)
{
	if (X%BLOQUE==0 && Y%BLOQUE==0 && T%BLOQUE==0) {
		kerd_stricte<<<dim3(KERD(Y, BLOQUE), KERD(T, BLOQUE)), dim3(BLOQUE, BLOQUE)>>>(
			_t_MODE, GRAINE,
			X_vars, Y_vars,
			X, Y,
			depart, T,
			DEPART_x,
			x, y,
			p,
			locd);
		ATTENDRE_CUDA();
	} else {
		nvidia_dot1d_conn_alea_naive(
			_t_MODE, GRAINE,
			X_vars, Y_vars,
			X, Y,
			depart, T,
			DEPART_x,
			x, y,
			p,
			locd);
	}
}

//	============================= Derivation ==============================

static __global__ void kerd_stricte_deriv(
	uint _t_MODE, uint GRAINE,
	uint X_vars, uint Y_vars,
	uint X, uint Y,
	uint depart, uint T,
	uint DEPART_x,
	float * x, float * y,
	float * p,
	float * locd,
	float * dy,
	float * dx,
	float * dp)
{
	// <KERD(T, BLOQUE), KERD(Y,BLOQUE)>
	// <         BLOQUE,         BLOQUE>

	__shared__ float __partage__x[BLOQUE][BLOQUE];
	__shared__ float __partage__p[BLOQUE][BLOQUE];

	__shared__ float __partage__dx[BLOQUE][BLOQUE];
	__shared__ float __partage__dp[BLOQUE][BLOQUE];

	uint thx = threadIdx.x;
	uint thy = threadIdx.y;

	uint _y = thx + blockIdx.x * blockDim.x;
	uint _t = thy + blockIdx.y * blockDim.y;

	float _locd = locd[(0+_t)*Y + _y] * dy[(0+_t)*Y + _y];

	atomicAdd(&dp[_y*(CONNECTIONS+1) + (CONNECTIONS+1-1)], _locd);

	FOR(0, d, X/BLOQUE) {
		__partage__x[thy][thx] = x[(0+_t)*(X_vars) + DEPART_x + POINTEUR(_y,d*BLOQUE + thx,X)];
		__partage__p[thy][thx] = p[_y*(CONNECTIONS+1) + d*BLOQUE + thy];
		__partage__dx[thy][thx] = 0;
		__partage__dp[thy][thx] = 0;
		__syncthreads();

	//#pragma unroll
		FOR(0, i, BLOQUE) {
			//s += __partage__x[thy][i] * __partage__p[i][thx];
			atomicAdd(&__partage__dx[thy][i], _locd * __partage__p[i][thx]);
			atomicAdd(&__partage__dp[i][thx], _locd * __partage__x[thy][i]);
		}
		__syncthreads();

		atomicAdd(&dx[(0+_t)*(X_vars) + DEPART_x + POINTEUR(_y, d*BLOQUE + thx,X)], __partage__dx[thy][thx]); 
		atomicAdd(&dp[_y*(CONNECTIONS+1) + d*BLOQUE + thy], __partage__dp[thy][thx]);
	}
};

void d_nvidia_dot1d_conn_alea_shared(
	uint _t_MODE, uint GRAINE,
	uint X_vars, uint Y_vars,
	uint X, uint Y,
	uint depart, uint T,
	uint DEPART_x,
	float * x, float * y,
	float * p,
	float * locd,
	float * dy,
	float * dx,
	float * dp)
{
	if (X%BLOQUE==0 && Y%BLOQUE==0 && T%BLOQUE==0) {
		kerd_stricte_deriv<<<dim3(KERD(Y, BLOQUE), KERD(T, BLOQUE)), dim3(BLOQUE, BLOQUE)>>>(
			_t_MODE, GRAINE,
			X_vars, Y_vars,
			X, Y,
			depart, T,
			DEPART_x,
			x, y,
			p,
			locd,
			dy,
			dx,
			dp);
		ATTENDRE_CUDA();
	} else {
		d_nvidia_dot1d_conn_alea_naive(
			_t_MODE, GRAINE,
			X_vars, Y_vars,
			X, Y,
			depart, T,
			DEPART_x,
			x, y,
			p,
			locd,
			dy,
			dx,
			dp);
	}
}