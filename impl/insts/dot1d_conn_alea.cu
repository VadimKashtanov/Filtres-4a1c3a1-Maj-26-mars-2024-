#include "hip/hip_runtime.h"
#include "dot1d_conn_alea.cuh"

#include "../../impl_tmpl/tmpl_etc.cu"

void cree_dot1d_conn_alea(Mdl_t * mdl, uint c)
{
	uint X=mdl->Y[c-1], Y=mdl->Y[c];
	mdl->inst_POIDS        [c] = (CONNECTIONS+1)*mdl->Y[c];
	mdl->inst_VARS         [c] = mdl->Y[c];
	mdl->inst_LOCDS        [c] = mdl->Y[c];
	mdl->inst_SORTIES      [c] = mdl->Y[c];
	mdl->inst_DEPART_SORTIE[c] = mdl->Y[c] - mdl->Y[c];
	//
	mdl->p[c] = alloc<float>(mdl->inst_POIDS[c]);

	FOR(0, y, Y) {
		FOR(0, x, CONNECTIONS+1) {
			mdl->p[c][y*(CONNECTIONS+1)+x] = (2*rnd()-1) * sqrtf(/*10.0*/ 6.0 / (X+Y));
		}
	}
};

void plume_dot1d_conn_alea(Mdl_t * mdl, uint c)
{
	printf("POIDS dot1d_conn_alea: \n");
	uint X=mdl->Y[c-1], Y=mdl->Y[c];
	FOR(0, y, Y) {
		printf("y=%i : ", y);
		FOR(0, x, CONNECTIONS) {
			printf("%+f,", mdl->p[c][y*(CONNECTIONS+1)+x]);
		}
		printf(" biais=%+f\n", mdl->p[c][y*(CONNECTIONS+1)+CONNECTIONS+1-1]);
	}
};

void intel_dot1d_conn_alea(
	uint _t_MODE, uint GRAINE,
	uint X_vars, uint Y_vars,
	uint X, uint Y,
	uint depart, uint T,
	uint DEPART_x,
	float * x, float * y,
	float * p,
	float * locd)
{
/*#pragma omp parallel
#pragma omp for*/
	FOR(0, t, T) {
		FOR(0, _y, Y) {
			float s = p[_y*(CONNECTIONS+1)+(CONNECTIONS+1-1)];
			FOR(0, k, CONNECTIONS) {
				float __x = x[(0+t)*X_vars+DEPART_x+POINTEUR(_y,k,X)];
				float __p = p[_y*(CONNECTIONS+1)+k];
				s += __x * __p;
			}
			float a = dot1d_conn_alea_ACTIV(dot1d_conn_alea_ACTIVATION, s);
			y[(0+t)*Y+_y]    = a;
			locd[(0+t)*Y+_y] = dot1d_conn_alea_dACTIV(dot1d_conn_alea_ACTIVATION, s, a);
		}
	}
}

void d_intel_dot1d_conn_alea(
	uint _t_MODE, uint GRAINE,
	uint X_vars, uint Y_vars,
	uint X, uint Y,
	uint depart, uint T,
	uint DEPART_x,
	float * x, float * y,
	float * p,
	float * locd,
	float * dy,
	float * dx,
	float * dp)
{
//#pragma omp parallel
//#pragma omp for
/*	FOR(0, t, T) {
		FOR(0, _y, Y) {
			float _locd = locd[(depart+t)*Y+_y] * dy[(depart+t)*Y+_y];
			dp[_y*(X+1)+(X+1-1)] += _locd;
			FOR(0, k, X) {
				//s += x[t*X+k] * p[y*(X+1)+k];
				dx[(depart+t)*X+k]      += _locd * p[_y*(X+1)+k];
				dp[_y*(X+1)+k] += _locd * x[(depart+t)*X+k];
			}
		}
	}*/

	//dx = (p @ ((y-_y)*dtanh(x@p)).T).T
/*#pragma omp parallel
#pragma omp for*/
	FOR(0, t, T) {
		FOR(0, _x, X) {
			//float _locd = locd[(depart+t)*Y+_y] * dy[(depart+t)*Y+_y];
			float s = 0;
			FOR(0, k, Y) {
				int lui_est_connectee = -1;	//lui est connecté par un poid, qui sera de `lui_est_connectee` addresse
				FOR(0, i, CONNECTIONS) {
					if (POINTEUR(k,i,X) == _x) {
						lui_est_connectee = (int)i;
						float __x = p[k*(CONNECTIONS+1)+lui_est_connectee];//x[(depart+t)*X+k];
						float __p = locd[(depart+t)*Y+k] * dy[(depart+t)*Y+k];//p[_y*(X+1)+k];
						s += __x * __p;
					}
				}
			}
			dx[(depart+t)*X_vars+DEPART_x+_x]  += s;
		}
	}

	//dp = x.T @ ((y-_y)*dtanh(x@p))
/*#pragma omp parallel
#pragma omp for*/
	FOR(0, _y, Y) {
		float dbiais = 0;
		FOR(0, _x, CONNECTIONS) {
			float s = 0;
			FOR(0, t, T) {
				float __x = locd[(depart+t)*Y+_y] * dy[(depart+t)*Y+_y];//x[(depart+t)*X+k];
				float __p = x[(depart+t)*X_vars+DEPART_x+POINTEUR(_y,_x,X)];//p[_y*(X+1)+k];
				s += __x * __p;
				if (_x == 0) {	//	Biais
					dbiais += __x;
				}
			}
			dp[_y*(CONNECTIONS+1)+_x] += s;
		}
		dp[_y*(CONNECTIONS+1)+(CONNECTIONS+1-1)] += dbiais;
	}
}

//	=========================================================

void f_dot1d_conn_alea(Mdl_t * mdl, uint inst, uint mode, uint t0, uint t1, uint _t_MODE, uint GRAINE) {
	uint X=mdl->Y[inst-1], Y=mdl->Y[inst];
	uint X_vars=mdl->inst_VARS[inst-1], Y_vars=mdl->inst_VARS[inst];
	uint depart = 0;//t0;
	uint T = (t1-t0);
	ASSERT(T == mdl->T);
	uint DEPART_x = mdl->inst_DEPART_SORTIE[inst-1];
	if (mode == 0) {
		intel_dot1d_conn_alea(
			_t_MODE, GRAINE,
			X_vars, Y_vars,
			X, Y,
			depart, T,
			DEPART_x,
			mdl->y[inst-1], mdl->y[inst],
			mdl->p[inst],
			mdl->l[inst]);
	} else if (mode == 1) {
		nvidia_dot1d_conn_alea_naive(
			_t_MODE, GRAINE,
			X_vars, Y_vars,
			X, Y,
			depart, T,
			DEPART_x,
			mdl->y__d[inst-1], mdl->y__d[inst],
			mdl->p__d[inst],
			mdl->l__d[inst]);
	} else if (mode == 2) {
		nvidia_dot1d_conn_alea_shared(
			_t_MODE, GRAINE,
			X_vars, Y_vars,
			X, Y,
			depart, T,
			DEPART_x,
			mdl->y__d[inst-1], mdl->y__d[inst],
			mdl->p__d[inst],
			mdl->l__d[inst]);
	} else if (mode == 3) {
		nvidia_dot1d_conn_alea_shared_2_16(
			_t_MODE, GRAINE,
			X_vars, Y_vars,
			X, Y,
			depart, T,
			DEPART_x,
			mdl->y__d[inst-1], mdl->y__d[inst],
			mdl->p__d[inst],
			mdl->l__d[inst]);
	} else {
		ERR("Pas de mode %i pour cuda f(x)", mode);
	}
}

//	----------------------------

void df_dot1d_conn_alea(Mdl_t * mdl, uint inst, uint mode, uint t0, uint t1, uint _t_MODE, uint GRAINE) {
	uint X=mdl->Y[inst-1], Y=mdl->Y[inst];
	uint X_vars=mdl->inst_VARS[inst-1], Y_vars=mdl->inst_VARS[inst];
	uint depart = 0;//t0;
	uint T = (t1-t0);
	ASSERT(T == mdl->T);
	uint DEPART_x = mdl->inst_DEPART_SORTIE[inst-1];
	if (mode == 0) {
		d_intel_dot1d_conn_alea(
			_t_MODE, GRAINE,
			X_vars, Y_vars,
			X, Y,
			depart, T,
			DEPART_x,
			mdl->y[inst-1], mdl->y[inst],
			mdl->p[inst],
			mdl->l[inst],
			mdl->dy[inst],
			mdl->dy[inst-1],
			mdl->dp[inst]);
	} else if (mode == 1) {
		d_nvidia_dot1d_conn_alea_naive(
			_t_MODE, GRAINE,
			X_vars, Y_vars,
			X, Y,
			depart, T,
			DEPART_x,
			mdl->y__d[inst-1], mdl->y__d[inst],
			mdl->p__d[inst],
			mdl->l__d[inst],
			mdl->dy__d[inst],
			mdl->dy__d[inst-1],
			mdl->dp__d[inst]);
	} else if (mode == 2) {
		d_nvidia_dot1d_conn_alea_shared(
			_t_MODE, GRAINE,
			X_vars, Y_vars,
			X, Y,
			depart, T,
			DEPART_x,
			mdl->y__d[inst-1], mdl->y__d[inst],
			mdl->p__d[inst],
			mdl->l__d[inst],
			mdl->dy__d[inst],
			mdl->dy__d[inst-1],
			mdl->dp__d[inst]);
	} else if (mode == 3) {
		d_nvidia_dot1d_conn_alea_shared_2_16(
			_t_MODE, GRAINE,
			X_vars, Y_vars,
			X, Y,
			depart, T,
			DEPART_x,
			mdl->y__d[inst-1], mdl->y__d[inst],
			mdl->p__d[inst],
			mdl->l__d[inst],
			mdl->dy__d[inst],
			mdl->dy__d[inst-1],
			mdl->dp__d[inst]);
	} else {
		ERR("Pas de mode %i pour cuda df(x)", mode);
	}
}