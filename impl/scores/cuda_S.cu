#include "hip/hip_runtime.h"
#include "S.cuh"

#include "../../impl_tmpl/tmpl_etc.cu"

#define pseudo_alea_d_une_grain(i) ((float)((121+(i%1234))*31 % 1001 ) / 1001.0)

//	===============================================================

static __global__ void kerd_nvidia_score_somme_USD_NORME(
	uint _t_MODE, uint GRAINE,
	float * y, uint depart, uint T,
	float * score, float * _PRIXS)
{
	//uint t = threadIdx.x + blockIdx.x + blockDim.x;
	float usd = 0.01;
	float delta;
	float max_delta, min_delta;
	FOR(0, t, T) {
		uint cuda_depart_plus_t = t_MODE_GENERALE(_t_MODE, GRAINE, depart, DEPART, FIN, t);

		float s = 0;
		float _y = y[(0 + t)*P + 0];
		float alea = 2*pseudo_alea_d_une_grain(t + ((uint)_y % 10001))-1;
		_y += alea * SCORE_Y_COEF_BRUIT;
		//
		delta = usd * _y * (_PRIXS[cuda_depart_plus_t+1] / _PRIXS[cuda_depart_plus_t] -1);
		usd += delta;
		if (t == 0) {
			max_delta = fabs(delta);
			min_delta = fabs(delta);
		}
		//
		if (usd < 0) usd = 0;
		if (delta > max_delta) max_delta = fabs(delta);
		if (delta < min_delta) min_delta = fabs(delta);
	}
	usd = 0.01;
	FOR(0, t, T) {
		uint cuda_depart_plus_t = t_MODE_GENERALE(_t_MODE, GRAINE, depart, DEPART, FIN, t);

		float s = 0;
		//FOR(0, p, P) {
		uint p = 0;
			float _y = y[(0 + t)*P + p];
			float alea = 2*pseudo_alea_d_une_grain(t + ((uint)_y % 10001))-1;
			_y += alea * SCORE_Y_COEF_BRUIT;
			//
			float delta = usd * _y * (_PRIXS[cuda_depart_plus_t+1] / _PRIXS[cuda_depart_plus_t] -1);
			usd += delta;
			//
			float importance = (/*usd*/fabs(delta) - min_delta)/(max_delta-min_delta);
			//
			s += importance*(P-p)*cuda_SCORE(
				_y, _PRIXS[cuda_depart_plus_t+p+1], _PRIXS[cuda_depart_plus_t], alea * SCORE_Y_COEF_BRUIT
			);
		//}
		atomicAdd(score, s);
	}
};

static __global__ void kerd_nvidia_score_somme(
	uint _t_MODE, uint GRAINE,
	float * y, uint depart, uint T,
	float * score, float * _PRIXS)
{
	uint t = threadIdx.x + blockIdx.x + blockDim.x;
	if (t < T) {
		uint cuda_depart_plus_t = t_MODE_GENERALE(_t_MODE, GRAINE, depart, DEPART, FIN, t);

		float s = 0;
		FOR(0, p, P) {
			float _y = y[(0 + t)*P + p];
			float alea = 2*pseudo_alea_d_une_grain(t + ((uint)_y % 10001))-1;
			_y += alea * SCORE_Y_COEF_BRUIT;
			s += (P-p)*cuda_SCORE(
				_y, _PRIXS[/*depart+t*/cuda_depart_plus_t+p+1], _PRIXS[/*depart+t*/cuda_depart_plus_t], alea * SCORE_Y_COEF_BRUIT
			);
		}
		atomicAdd(score, s);
	}
};

float nvidia_somme_score(float * y, uint depart, uint T, uint _t_MODE, uint GRAINE)
{
	float * somme_score__d = cudalloc<float>(1);
	CONTROLE_CUDA(hipMemset(somme_score__d, 0, sizeof(float)*1));
	if (SCORE_AVEC_USD_NORME == false) {
		kerd_nvidia_score_somme<<<dim3(KERD(T,1)),dim3(1)>>>(
			_t_MODE, GRAINE,
			y, depart, T,
			somme_score__d, prixs__d
		);
	} else {
		kerd_nvidia_score_somme_USD_NORME<<<dim3(1),dim3(1)>>>(
			_t_MODE, GRAINE,
			y, depart, T,
			somme_score__d, prixs__d
		);
	}
	ATTENDRE_CUDA();
	float somme_score;
	CONTROLE_CUDA(hipMemcpy(&somme_score, somme_score__d, sizeof(float)*1, hipMemcpyDeviceToHost));
	CONTROLE_CUDA(hipFree(somme_score__d));
	return somme_score;
};

float  nvidia_score_finale(float somme, uint T, uint _t_MODE, uint GRAINE) {
	return APRES_SCORE(somme / (float)(P * T));
};

/*	Regularisation L2
	Pas oublier le Attention mechanisme
*/

//	===============================================================

float d_nvidia_score_finale(float somme, uint T, uint _t_MODE, uint GRAINE) {
	return dAPRES_SCORE(somme / (float)(P * T)) / (float)(P * T);
};

//	===============================================================

static __global__ void kerd_nvidia_score_dpowf_USD_NORME(
	uint _t_MODE, uint GRAINE,
	float _dy, float * y, float * dy,
	uint depart, uint T,
	float * _PRIXS)
{
	float usd = 0.01;
	float delta;
	float max_delta, min_delta;
	FOR(0, t, T) {
		uint cuda_depart_plus_t = t_MODE_GENERALE(_t_MODE, GRAINE, depart, DEPART, FIN, t);

		float s = 0;
		float _y = y[(0 + t)*P + 0];
		float alea = 2*pseudo_alea_d_une_grain(t + ((uint)_y % 10001))-1;
		_y += alea * SCORE_Y_COEF_BRUIT;
		//
		delta = usd * _y * (_PRIXS[cuda_depart_plus_t+1] / _PRIXS[cuda_depart_plus_t] -1);
		usd += delta;
		if (t == 0) {
			max_delta = fabs(delta);
			min_delta = fabs(delta);
		}
		//
		if (usd < 0) usd = 0;
		if (delta > max_delta) max_delta = fabs(delta);
		if (delta < min_delta) min_delta = fabs(delta);
	}
	usd = 0.01;
	FOR(0, _t, T) {
		uint cuda_depart_plus_t = t_MODE_GENERALE(_t_MODE, GRAINE, depart, DEPART, FIN, _t);
		//FOR(0, p, P) {
		uint p = 0;
		float _y = y[(0+_t)*P+p];
		float alea = 2*pseudo_alea_d_une_grain(_t + ((uint)_y % 10001))-1;
		//
		_y += alea*SCORE_Y_COEF_BRUIT;
		//
		float delta = usd * _y * (_PRIXS[cuda_depart_plus_t+1] / _PRIXS[cuda_depart_plus_t] -1);
		usd += delta;
		//
		float importance = (/*usd*/fabs(delta) - min_delta)/(max_delta-min_delta);
		//
		dy[(0+_t)*P+p] = _dy * importance * (P-p)*cuda_dSCORE(
			_y, _PRIXS[cuda_depart_plus_t+p+1], _PRIXS[cuda_depart_plus_t], alea*SCORE_Y_COEF_BRUIT
		);
		//}
	}
};

static __global__ void kerd_nvidia_score_dpowf(
	uint _t_MODE, uint GRAINE,
	float _dy, float * y, float * dy,
	uint depart, uint T,
	float * _PRIXS)
{
	uint _t = threadIdx.x + blockIdx.x * blockDim.x;

	if (_t < T) {
		uint cuda_depart_plus_t = t_MODE_GENERALE(_t_MODE, GRAINE, depart, DEPART, FIN, _t);
		FOR(0, p, P) {
			float _y = y[(0+_t)*P+p];
			float alea = 2*pseudo_alea_d_une_grain(_t + ((uint)_y % 10001))-1;
			dy[(0+_t)*P+p] = _dy * (P-p)*cuda_dSCORE(
				y[(0+_t)*P+p]+alea*SCORE_Y_COEF_BRUIT, _PRIXS[/*depart+_t*/cuda_depart_plus_t+p+1], _PRIXS[/*depart+_t*/cuda_depart_plus_t/*+p*/], alea*SCORE_Y_COEF_BRUIT
			);
		}
	}
};

void d_nvidia_somme_score(float d_score, float * y, float * dy, uint depart, uint T, uint _t_MODE, uint GRAINE) {
	if (SCORE_AVEC_USD_NORME == false) {
		kerd_nvidia_score_dpowf<<<dim3(KERD(T,1024)), dim3(1024)>>>(
			_t_MODE, GRAINE,
			d_score,
			y, dy,
			depart, T,
			prixs__d
		);
	} else if (SCORE_AVEC_USD_NORME == true) {
		kerd_nvidia_score_dpowf_USD_NORME<<<dim3(1), dim3(1)>>>(
			_t_MODE, GRAINE,
			d_score,
			y, dy,
			depart, T,
			prixs__d
		);
	}
	ATTENDRE_CUDA();
};